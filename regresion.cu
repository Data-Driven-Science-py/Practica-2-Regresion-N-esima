#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <regresion.cuh>

template<unsigned int N, unsigned int samples>
Regression<N, samples>::Regression (float* x[N*samples], float* y[samples]) : x(x), y(y) {};

template<unsigned int N, unsigned int samples>
void Regression<N, samples>::runtime (void) {
    firstKernel<<<samples, N>>>(void);
    secondKernel<<<samples, N>>>(void);
};

template<unsigned int N, unsigned int samples>
__device__ void Regression<N, samples>::prefetch_data (void) {
};

template<unsigned int N, unsigned int samples>
__device__ void Regression<N, samples>::grad_step (void) const {
};

template<unsigned int N, unsigned int samples>
__device__ void Regression<N, samples>::forward (void) const {
    // mult -> atomicSum -> fetch after prefetch of the next kernel -> grad_step

};

// Make the forward step
template<unsigned int N, unsigned int samples>
__global__ void Regression<N, samples>::firstKernel (void) {
};

// prefetch data and bring final data to device
template<unsigned int N, unsigned int samples>
__global__ void Regression<N, samples>::secondKernel (void) {
    prefetch_data(void);
    // put the semafor
    // fetch the atomic data
    // make the grad_step
};
